
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <math.h>
#include <chrono>
#include <iomanip>
#include <fstream>
using namespace std;
using namespace std::chrono;
typedef unsigned long long ULL;
ofstream primeresult;
ofstream timeresult;

__global__ void cuda_erastothenes_sieve (ULL *marked, ULL *limit, ULL *n, int *totalThreads) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("index %d,limit %llu , n %llu , totalThreads %d \n",index, *limit , *n , *totalThreads);
    //printf("block %d, thread %d \n", blockIdx.x, threadIdx.x);
    marked[0]=1;
    marked[1]=1;
    index=index+2;
    if(*totalThreads>*n || index >*limit){
        return ;
    }else if(*totalThreads ==1){
        for(ULL p=2;p<=*limit;p++){
            for(ULL multiple=2*p; multiple<*n; multiple+=p){
                marked[multiple]=1;
            }    
        }
    }
    else{
        //printf("index %d,limit %llu , n %llu , totalThreads %d \n",index, *limit , *n , *totalThreads);
        for(ULL p=index;p<=*limit;p+=*totalThreads){
                if(marked[p]==1 || p%2==0 && p>2) return;
                //printf("index %d , p%llu \n",index,p);
                for(ULL multiple=2*p; multiple<*n; multiple+=p){
                    //printf("multiple %llu \n",multiple);
                    marked[multiple]=1;
                }
        }
    }
    __syncthreads();
}


int main(int argc, char **argv){
    ULL size;
    ULL *list,*d_list,limit,*d_limit,*d_n,n;
    n=atoi(argv[1]);
    int threads=atoi(argv[2]);
    //int thread_per_block=atoi(argv[3]);
    int blocks= floor(n/threads);
    int totalThreads = threads;
    int *d_totalThreads; 
    primeresult.open("cudaSieveList.txt");
    timeresult.open("cudaSieveResult.txt",ios::out | ios::app );

    size = n*sizeof(ULL);
    limit = floor(sqrt(n));

    
    hipMalloc((void**)&d_list,size);
    hipMalloc((void**)&d_limit,sizeof(ULL));
    hipMalloc((void**)&d_n,sizeof(ULL));
    hipMalloc((void**)&d_totalThreads,sizeof(int));
    
    list = (ULL*) malloc(size);
 
    memset(list,0,size);
    hipMemset(d_list, 0, size);

    hipMemcpy(d_list           ,list           ,size           ,hipMemcpyHostToDevice);
    hipMemcpy(d_limit          ,&limit         ,sizeof(ULL)    ,hipMemcpyHostToDevice);
    hipMemcpy(d_n              ,&n             ,sizeof(ULL)    ,hipMemcpyHostToDevice);
    hipMemcpy(d_totalThreads   ,&totalThreads  ,sizeof(int)    ,hipMemcpyHostToDevice);


    auto begin= std::chrono::high_resolution_clock::now();
    cuda_erastothenes_sieve<<<blocks,threads>>>(d_list,d_limit,d_n,d_totalThreads);
    auto end = std::chrono::high_resolution_clock::now();

    auto duration = duration_cast<std::chrono::microseconds>(end - begin);
    
    timeresult<<duration.count()<<endl;

    hipMemcpy(list,d_list, size, hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      return 0;
    }
    
   for(ULL i=0;i<n;i++){
       if(list[i]==0){
            primeresult<<i<<endl;
            //cout<<i<<endl;
        };
    } 


    
    //cout<<duration.count()<<endl;

    free(list);
    hipFree(d_list);
    hipFree(d_limit);
    hipFree(d_n);
    primeresult.close();
    timeresult.close();
    return 0;
}